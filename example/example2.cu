
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void example(float *x, float *y, int nfloat)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float d;
  float tmp;
  int   i=0; // simplify example, removing 'for'

  tmp = x[i] / (x[i]-y[i]);
  d = sqrt(tmp);

  tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {
    printf("dot: %f\n", d);
  }
}

int main(int argc, char **argv)
{
  int nfloat = 2;
  int nbytes = nfloat*sizeof(float);
  float *d_a = 0;
  hipMalloc(&d_a, nbytes);

  float *data = (float *)malloc(nbytes);
  for (int i=0; i < nfloat; ++i)
  {
    data[i] = (float)(i+1);
  }

  hipMemcpy((void *)d_a, (void *)data, nbytes, hipMemcpyHostToDevice);
  printf("Calling kernel\n");

  example<<<1,1>>>(d_a, d_a, nfloat);

  hipDeviceSynchronize();
  printf("done\n");

  return 0;
}
